#include <hip/hip_runtime.h>
//#include <cublas_v2.h>
//#include <cublasXt.h>
//#include <cudnn.h>
//#include <nccl.h>

#include <cassert>
#include <chrono>
#include <iostream>

#define CUDA_CHECK(e) (assert(hipSuccess == (e)))
#define CUBLAS_CHECK(e) (assert(HIPBLAS_STATUS_SUCCESS == (e)))
#define CUDNN_CHECK(e) (assert(CUDNN_STATUS_SUCCESS == (e)))

class AtomicReduceMap {
public:
  template <typename T>
  __device__ __forceinline__ static void Op(T& dst, const T& src);
};

template <>
__device__ __forceinline__
void AtomicReduceMap::Op<float>(float& dst, const float& src) {
  (void)atomicAdd(&dst, src);
}

template <typename T, typename Map>
__global__ void reduce(
    uint32_t buf_size,
    const T* x,
    T* y)
{
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < buf_size) {
    Map::template Op<T>(y[idx], x[idx]);
  }
}

int main(int argc, const char** argv) {
  int num_devices = 0;
  CUDA_CHECK(hipGetDeviceCount(&num_devices));
  std::clog << "num devices: " << num_devices << std::endl;
  num_devices = 1;

  const size_t buf_size = 64UL * 1024UL * 1024UL;

  hipStream_t stream = NULL;
  float* x = NULL;
  float* y = NULL;

  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipStreamCreate(&stream));
  CUDA_CHECK(hipMalloc((void**)&x, buf_size * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&y, buf_size * sizeof(float)));
  CUDA_CHECK(hipStreamSynchronize(stream));
  CUDA_CHECK(hipDeviceSynchronize());

  const int num_trials = 1000;
  double avg_elapsed_ms = 0.0;
  std::clog << "running: reduce" << std::endl;

  for (int t = 0; t < num_trials; ++t) {
    auto start = std::chrono::steady_clock::now();

    CUDA_CHECK(hipSetDevice(0));
    reduce<float, AtomicReduceMap><<<(buf_size+1024-1)/1024, 1024, 0, stream>>>(
        buf_size, x, y);
    CUDA_CHECK(hipStreamSynchronize(stream));

    auto lap = std::chrono::steady_clock::now();
    auto diff = lap - start;
    avg_elapsed_ms += std::chrono::duration<double, std::milli>(diff).count();
  }

  avg_elapsed_ms /= num_trials;
  double avg_bandwidth = ((double)(buf_size * sizeof(float)) * 1.0e-9) / (avg_elapsed_ms * 1.0e-3);
  std::clog << "  avg wallclock:  " << avg_elapsed_ms << " ms" << std::endl;
  std::clog << "  avg bandwidth:  " << avg_bandwidth << " GB/s" << std::endl;

  return 0;
}
